#include "hip/hip_runtime.h"
#include "header.h"
#include "dijkstra.cu"

int main(int argc, char *argv[]) {
    //variable
    int num_of_thread = atoi(argv[1]);
    int num_of_vertices = strtol(argv[2],NULL,10);
    size_t size = num_of_vertices * num_of_vertices * sizeof(int);
    int *adj_matrix, *result_matrix;
    int *dev_matrix, *dev_result;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threads_per_block = num_of_thread;
    int blocks_in_grid = ceil(float(num_of_vertices)/threads_per_block);
    // dim3 threads_per_block(num_of_thread, num_of_thread, 1);
    // dim3 blocks_in_grid(ceil( float(num_of_vertices) / threads_per_block.x ),
    //                     ceil( float(num_of_vertices) / threads_per_block.y ), 1 );
    // Allocate memory on host
    result_matrix = (int*)malloc(size);
    adj_matrix = (int*)malloc(size);
    
    // Allocate memory on device
    hipMalloc(&dev_matrix, size);
    hipMalloc(&dev_result, size);

    // Random matrix
    randMatriks(adj_matrix, num_of_vertices);
    // randMatriks(result_matrix, num_of_vertices);
    PrintMatriks(result_matrix, num_of_vertices);

    // printf("~=== Awal  ===~\n");
    PrintMatriks(adj_matrix, num_of_vertices);

    // Copy data from host to device
    hipMemcpy(dev_matrix, adj_matrix, size, hipMemcpyHostToDevice);
    
    // start
    hipEventRecord(start);

    solution<<< blocks_in_grid, threads_per_block >>>(dev_matrix, dev_result, num_of_vertices);
    hipDeviceSynchronize();

    // Copy data from device to host
    hipMemcpy(result_matrix, dev_result, size, hipMemcpyDeviceToHost);

    // stop
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("\n");
    printf("~=== Hasil Matriks===~\n");
    PrintMatriks(result_matrix, num_of_vertices);
    printf("\ntime: %f microsecond(s)\n", milliseconds*1000);
    
    // free host memory
    free(adj_matrix);
    free(result_matrix);
    
    // free device memory
    hipFree(dev_matrix);
    hipFree(dev_result);

    return 0;
}