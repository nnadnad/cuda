
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define true 1
#define false 0


__device__
int min_distance(int dist[], int spt_set[], int n) {
    int min = INT_MAX, min_index;
    for (int v = 0; v < n; v++) {
        if (spt_set[v] == false && dist[v] <= min){
            min = dist[v], min_index = v;
        }
    }
    return min_index;
}

__device__
void dijkstra(int* graph, int* res, int src, int n) {
    // inisiasi ukuran matriks maksimal yang dibutuhkan sesuai testcase
    int dist[3000];
    int spt_set[3000];
    

    //init distance and spt_set
    for (int i = 0; i < n; i++) {
        dist[i] = INT_MAX;
        spt_set[i] = false;
    }

    // init distance dengan 0 semua
    dist[src] = 0;

    for (int count = 0; count < n - 1; count++) {
        int u = min_distance(dist, spt_set, n);
        spt_set[u] = true;
      for (int v = 0; v < n; v++) {
        if (!spt_set[v] && graph[u*n+v] && dist[u] != INT_MAX && dist[u] + graph[u*n+v] < dist[v]) {
            dist[v] = dist[u] + graph[u*n+v];
        }
      }
    }
    for (int i = 0; i < n; i++) {
        res[src*n + i] = dist[i];
    }
}

// random matriks dengan nim
__host__
void random_matriks(int* host_matrix, int num_nodes) {
    srand(13517074);
    // init distance
    for (int i = 0; i < num_nodes; i++) {
        for (int j = i; j < num_nodes; j++) {
            if (i == j) {
                host_matrix[i*num_nodes + j] = 0;
            } else {
                host_matrix[i*num_nodes + j] = rand() % 100;
                host_matrix[j*num_nodes + i] = host_matrix[i*num_nodes + j];
            }
        }
    }
}

__host__
void print_matriks(int* host_matrix, int num_nodes) {
    for (int i = 0; i < num_nodes; i++) {
        for (int j = 0; j < num_nodes; j++) {
            printf("%d\t", host_matrix[i*num_nodes + j]);
        }
        printf("\n");
    }
}

__global__
void solution (int* graph, int* result, int nodes_count) {
  // init source for check matriks
  /* gridDim: variabel yang berisi dimensi dari grid.
   * blockIdx: variabel yang berisi index block di mana thread ini berada.
   * blockDim: variabel yang berisi dimensi dari block.
   * threadIdx: variabel yang berisi index thread di dalam block. (untuk membedakan thread yang berada di block yang berbeda, gunakan blockIdx).*/
  int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nodes_count) {
        dijkstra(graph, result, i, nodes_count);
    }
}

int main(int argc, char *argv[]) {
  int nodes_count = strtol(argv[2], NULL, 10);
  int num_thread = atoi(argv[1]);
  size_t size = nodes_count*nodes_count*sizeof(int);


  //cuda variabel for calculate time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  // built in vaaribels
  int threads_per_block = num_thread;
  int blocks_in_grid = (nodes_count / threads_per_block) + 1;

  // Allocate memory on host
  int *host_matrix, *host_result_matrix;
  host_result_matrix = (int*)malloc(size);
  host_matrix = (int*)malloc(size);
  
  // Allocate memory on device
  int *device_matrix, *device_result_matrix;
  // check when the matrix allocation with the given size, it should not give an error
  hipError_t err = hipMalloc(&device_matrix, size);
  if(err != hipSuccess) {
    printf("Error Device Matrix: %s\n", hipGetErrorString(err));
  }
  err = hipMalloc(&device_result_matrix, size);
  if(err != hipSuccess) {
    printf("Error Device Result Matrix: %s\n", hipGetErrorString(err));
  }
  
  // Random matrix
  random_matriks(host_matrix, nodes_count);
  // Copy data from host to device
  hipMemcpy(device_matrix, host_matrix, size, hipMemcpyHostToDevice);

  // start calucate and time
  hipEventRecord(start);
  // run solution to find dijkstra
  solution<<< blocks_in_grid, threads_per_block >>>(device_matrix, device_result_matrix, nodes_count);
  hipError_t errAsync = hipDeviceSynchronize();
  if(errAsync != hipSuccess) {
    printf("Error Async: %s\n", hipGetErrorString(errAsync));
  }
  // Copy data from device to host
  hipMemcpy(host_result_matrix, device_result_matrix, size, hipMemcpyDeviceToHost);
  // stop
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // hasil matriks dan waktu kalkuklasi nya
  printf("\n");
  printf("~=== Result Matrix ===~\n");
  print_matriks(host_result_matrix, nodes_count);
  printf("\ntime execution: %f microsecond(s)\n", milliseconds*1000);
  
  // free host memory
  free(host_matrix);
  free(host_result_matrix);
  
  // free device memory
  hipFree(device_matrix);
  hipFree(device_result_matrix);

  return 0;
}
