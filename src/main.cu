
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define true 1
#define false 0


__device__
int minDistance(int dist[], int sptSet[], int num_vertex) {
  int min = INT_MAX, min_index;
  for (int v = 0; v < num_vertex; v++)
    if (sptSet[v] == false && dist[v] <= min)
      min = dist[v], min_index = v;
  return min_index;
}

__device__
void dijkstra(int* graph, int* res, int src, int num_vertex) {
  int dist[3000];
  int sptSet[3000];
  for (int i = 0; i < num_vertex; i++) 
    dist[i] = INT_MAX, sptSet[i] = false;

  dist[src] = 0;

  for (int count = 0; count < num_vertex - 1; count++) {
    int u = minDistance(dist, sptSet, num_vertex);

    sptSet[u] = true;

    for (int v = 0; v < num_vertex; v++)

      if (!sptSet[v] && graph[u*num_vertex+v] && dist[u] != INT_MAX
        && dist[u] + graph[u*num_vertex+v] < dist[v])
        dist[v] = dist[u] + graph[u*num_vertex+v];
  }

  for (int i = 0; i < num_vertex; i++)
    res[src*num_vertex + i] = dist[i];
}

__host__
void RandomMatrix(int* adj_matrix, int num_nodes) {
  srand(13517001);
  // init distance
  for (int i = 0; i < num_nodes; i++) {
    for (int j = i; j < num_nodes; j++) {
      if (i == j) {
        adj_matrix[i*num_nodes + j] = 0;
      } else {
          adj_matrix[i*num_nodes + j] = rand() % 100;
      adj_matrix[j*num_nodes + i] = adj_matrix[i*num_nodes + j];
      }
    }
  }
}

__host__
void PrintMatrix(int* adj_matrix, int num_nodes) {
  for (int i = 0; i < num_nodes; i++) {
    for (int j = 0; j < num_nodes; j++) 
      printf("%d\t", adj_matrix[i*num_nodes + j]);
    printf("\n");
  }
}

__global__
void debug(int x) {
  printf("DEBUG %d >>>\n", x);
}

__global__
void CalcDijkstra (int* graph, int* result, int nodes_count) {
  
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < nodes_count)
    dijkstra(graph, result, i, nodes_count);
}

int main(int argc, char *argv[]) {
  int nodes_count = strtol(argv[2], NULL, 10);
  int num_thread = atoi(argv[1]);
  size_t size = nodes_count*nodes_count*sizeof(int);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int threads_per_block = num_thread;
  int blocks_in_grid = (nodes_count/threads_per_block) +1;

  // Allocate memory on host
  int *adj_matrix, *result_matrix;
  result_matrix = (int*)malloc(size);
  adj_matrix = (int*)malloc(size);
  
  // Allocate memory on device
  int *dev_matrix, *dev_result;
  hipError_t err = hipMalloc(&dev_matrix, size);
  if(err != hipSuccess) printf("Error Malloc 1: %s\n", hipGetErrorString(err));
  err = hipMalloc(&dev_result, size);
  if(err != hipSuccess) printf("Error Malloc 2: %s\n", hipGetErrorString(err));
  
  // Random matrix
  RandomMatrix(adj_matrix, nodes_count);
  
  printf("~=== Awal  ===~\n");
  PrintMatrix(adj_matrix, nodes_count);

  // Copy data from host to device
  hipMemcpy(dev_matrix, adj_matrix, size, hipMemcpyHostToDevice);
  
  // start
  hipEventRecord(start);

  CalcDijkstra<<< blocks_in_grid, threads_per_block >>>(dev_matrix, dev_result, nodes_count);
  hipError_t errAsync = hipDeviceSynchronize();
  if(errAsync != hipSuccess) printf("Error Async: %s\n", hipGetErrorString(errAsync));

  // Copy data from device to host
  hipMemcpy(result_matrix, dev_result, size, hipMemcpyDeviceToHost);

  // stop
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("\n");
  printf("~=== Hasil ===~\n");
  PrintMatrix(result_matrix, nodes_count);
  printf("\ntime: %f microsecond(s)\n", milliseconds*1000);
  
  // free host memory
  free(adj_matrix);
  free(result_matrix);
  
  // free device memory
  hipFree(dev_matrix);
  hipFree(dev_result);

  return 0;
}
