#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <limits.h>

#include "boolean.h"



/**
 * Get vertex index with minimum distance which not yet included
 * in spt_set
 * @param  dist    distance from origin vertex to vertex with that index
 * @param  spt_set a set denoting vertices included in spt_set
 * @param n number of vertices in the graph
 * @return         index of minimum distance not yet included in spt_set
 */
__device__ int min_distance_idx(long *dist, bool spt_set[], int n) {
	// Initialize min value 
    int min = INT_MAX, min_index; 
    for (int i = 0; i < n; i++) {
        if (spt_set[i] == false && dist[i] <= min) {
            min = dist[i];
            min_index = i;
        }
    } 
    return min_index; 
}

/**
 * generate a graph with n vertices
 * @param  n number of vertices
 * @param matriks matriks untuk isi hasil random angka
 */
__host__ void randMatriks(int *matriks, int n) {
    srand(13517074);

    // isi matriks dengan bilangan random
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) {
                matriks[i*n + j] = 0;
            } else {
                matriks[i*n + j] = matriks[j*n + i] = rand()%100;
            }
        }
    }
}

__host__ void PrintMatriks(int *matriks, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d\t", matriks[i*n + j]);
            printf("\n");
        }
    }
}

__device__ int dijkstra(int* graph, int src, int n) {
    // output array, contains shortest distance form src to every vertices
    int *dist = (int*)malloc(n*sizeof(int));
    
    // spt_set[i] is true if vertex i already included in the shortest path tree
    bool  spt_set[n];
    // int *spt_set = (int*)malloc(n*sizeof(int));
    
    // initialize dist and spt_set
    for (int i = 0; i < n; i++) {
        dist[i] = INT_MAX;
        spt_set[i] = false;
    }

    // init path searching
    dist[src] = 0;

    // find the shortest path for all vertices
    for (int i = 0; i < n - 1; i++) {

        // pick vertex with minimun distance from src
        // form spt_set not yet processed
        int processed_vertex = min_distance_idx(dist, spt_set, n);
        
        // mark vertex as processed
        spt_set[processed_vertex] = true;
        
        
        for (int v = 0; v < n; v++) {
            if (!spt_set[v] 
                && graph[processed_vertex*n + v] 
                && dist[processed_vertex] != INT_MAX
                && dist[processed_vertex] + graph[processed_vertex*n + v] < dist[v]) {
                    dist[v] = dist[processed_vertex] + graph[processed_vertex*n + v];
                }
        }
    }

    // save result
    int* result;
    result = (int*)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++) {
        result[i] = dist[i];
    }
    free(dist);
    free(spt_set);
    
    return result;
}

__global__ void solution(int *graph, int *result, int n) {
    int src = blockDim.x * blockIdx.x + threadIdx.x;

    if (src < n) {
        // alokasi memori
        int *matriks = (int*)malloc(n * sizeof(int));

        //hitung dijkstra
        matriks = dijkstra(graph, src, n);

        // calculate
        for (int i = 0; i < n; i++) {
            result[src*n + i] = matriks[i];
        }
        free(matriks);
    }
}


