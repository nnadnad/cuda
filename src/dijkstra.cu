
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define true 1
#define false 0


__device__
int min_distance(int dist[], int spt_set[], int n) {
    int min = INT_MAX, min_index;
    for (int v = 0; v < n; v++) {
        if (spt_set[v] == false && dist[v] <= min){
            min = dist[v], min_index = v;
        }
    }
    return min_index;
}

__device__
void dijkstra(int* graph, int* res, int src, int n) {
    // inisiasi ukuran matriks maksimal yang dibutuhkan sesuai testcase
    int dist[3000];
    int spt_set[3000];
    

    //init distance and spt_set
    for (int i = 0; i < n; i++) {
        dist[i] = INT_MAX;
        spt_set[i] = false;
    }

    // init distance dengan 0 semua
    dist[src] = 0;

    for (int count = 0; count < n - 1; count++) {
        int u = min_distance(dist, spt_set, n);
        spt_set[u] = true;
    for (int v = 0; v < n; v++) {
        if (!spt_set[v] && graph[u*n+v] && dist[u] != INT_MAX && dist[u] + graph[u*n+v] < dist[v]) {
            dist[v] = dist[u] + graph[u*n+v];
        }
    }
    }
    for (int i = 0; i < n; i++) {
        res[src*n + i] = dist[i];
    }
}

// random matriks dengan nim
__host__
void RandomMatrix(int* matriksAwal, int num_nodes) {
    srand(13517074);
    // init distance
    for (int i = 0; i < num_nodes; i++) {
        for (int j = i; j < num_nodes; j++) {
            if (i == j) {
                matriksAwal[i*num_nodes + j] = 0;
            } else {
                matriksAwal[i*num_nodes + j] = rand() % 100;
                matriksAwal[j*num_nodes + i] = matriksAwal[i*num_nodes + j];
            }
        }
    }
}

__host__
void PrintMatrix(int* matriksAwal, int num_nodes) {
    for (int i = 0; i < num_nodes; i++) {
        for (int j = 0; j < num_nodes; j++) {
            printf("%d\t", matriksAwal[i*num_nodes + j]);
            printf("\n");
        }
    }
}

__global__
void solution (int* graph, int* result, int nodes_count) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < nodes_count) {
        dijkstra(graph, result, i, nodes_count);
    }
}